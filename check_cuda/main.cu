
#include <hip/hip_runtime.h>
#include <iostream>

int
main()
{
    hipDeviceProp_t	dP;
    const auto		rc = hipGetDeviceProperties(&dP, 0);
    if (rc != hipSuccess)
    {
        const auto	error = hipGetLastError();
	std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

	return rc; /* Failure */
    }

    const auto	min_cc = 4.0;
    const auto	arch = 10*dP.major + dP.minor;
    if (0.1*arch < min_cc)
    {
	std::cerr << "Compute Capability " << min_cc
		  << " or higher required: " << dP.major << '.' << dP.minor
		  << " found."
		  << std::endl;

	return -1; /* Failure */
    }

    std::cout << arch;

    return 0;
}
