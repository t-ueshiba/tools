
#include <hip/hip_runtime.h>
#include <iostream>

int
main()
{
    hipDeviceProp_t	dP;
    const auto		rc = hipGetDeviceProperties(&dP, 0);
    if (rc != hipSuccess)
    {
        const auto	error = hipGetLastError();
	std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;

	return rc; /* Failure */
    }

    const float		min_cc = 3.0;
    if ((dP.major + 0.1*dP.minor) < min_cc)
    {
	std::cerr << "Compute Capability " << min_cc
		  << " or higher required: " << dP.major << '.' << dP.minor
		  << " found."
		  << std::endl;

	return 1; /* Failure */
    }

    std::cout << dP.major << dP.minor;

    return 0;
}
